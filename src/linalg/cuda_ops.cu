
#include <hip/hip_runtime.h>
#ifdef HAVE_CUDA
#include <cuda_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <cmath>
#include <vector>
#include <algorithm>

namespace cuda_ops {

__global__ void matvec_kernel(const float* A, const float* x, float* y, int m, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m) {
        float temp = 0.0f;
        for (int j = 0; j < n; ++j) {
            temp += A[row * n + j] * x[j];
        }
        y[row] = temp;
    }
}

void matvec(const std::vector<float>& mat,
            const std::vector<float>& vec,
            int m, int n,
            std::vector<float>& res) {
    float *d_A, *d_x, *d_y;

    cudaMalloc(&d_A, mat.size() * sizeof(float));
    cudaMalloc(&d_x, vec.size() * sizeof(float));
    cudaMalloc(&d_y, res.size() * sizeof(float));

    cudaMemcpy(d_A, mat.data(), mat.size() * sizeof(float), cudaMemcpyHostToDevice);
    cudaMemcpy(d_x, vec.data(), vec.size() * sizeof(float), cudaMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (m + blockSize - 1) / blockSize;
    matvec_kernel<<<gridSize, blockSize>>>(d_A, d_x, d_y, m, n);
    cudaDeviceSynchronize();

    cudaMemcpy(res.data(), d_y, res.size() * sizeof(float), cudaMemcpyDeviceToHost);

    cudaFree(d_A);
    cudaFree(d_x);
    cudaFree(d_y);
}

std::vector<float> vector_subtract(const std::vector<float>& a, const std::vector<float>& b) {
    thrust::device_vector<float> d_a = a;
    thrust::device_vector<float> d_b = b;
    thrust::device_vector<float> d_res(a.size());

    thrust::transform(d_a.begin(), d_a.end(), d_b.begin(), d_res.begin(), thrust::minus<float>());

    std::vector<float> res(a.size());
    thrust::copy(d_res.begin(), d_res.end(), res.begin());
    return res;
}

std::vector<float> scalar_vector_prod(const float& scalar, const std::vector<float>& vec) {
    thrust::device_vector<float> d_vec = vec;
    thrust::device_vector<float> d_res(vec.size());

    thrust::transform(d_vec.begin(), d_vec.end(), d_res.begin(),
                      [scalar] __device__ (float x) { return scalar * x; });

    std::vector<float> res(vec.size());
    thrust::copy(d_res.begin(), d_res.end(), res.begin());
    return res;
}

float dot(const std::vector<float>& a, const std::vector<float>& b) {
    thrust::device_vector<float> d_a = a;
    thrust::device_vector<float> d_b = b;

    return thrust::inner_product(d_a.begin(), d_a.end(), d_b.begin(), 0.0f);
}

float l1_norm(const std::vector<float>& vec) {
    thrust::device_vector<float> d_vec = vec;
    return thrust::transform_reduce(d_vec.begin(), d_vec.end(),
                                    [] __device__ (float x) { return fabsf(x); },
                                    0.0f, thrust::plus<float>());
}

float l2_norm(const std::vector<float>& vec) {
    thrust::device_vector<float> d_vec = vec;
    float sum_squares = thrust::transform_reduce(d_vec.begin(), d_vec.end(),
                                                 [] __device__ (float x) { return x * x; },
                                                 0.0f, thrust::plus<float>());
    return std::sqrt(sum_squares);
}

float inf_norm(const std::vector<float>& vec) {
    thrust::device_vector<float> d_vec = vec;
    return thrust::transform_reduce(d_vec.begin(), d_vec.end(),
                                    [] __device__ (float x) { return fabsf(x); },
                                    0.0f, thrust::maximum<float>());
}

std::vector<float> l1_norm_projection(std::vector<float> vec, float tau) {
    return vec;
}

}
#endif